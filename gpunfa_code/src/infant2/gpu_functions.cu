#include "hip/hip_runtime.h"
// Original work:
// Copyright (C) 2010  
// Pierluigi Rolando (pierluigi.rolando@polito.it)
// Netgroup - DAUIN - Politecnico di Torino
//
// Niccolo' Cascarano (niccolo.cascarano@polito.it)
// Netgroup - DAUIN - Politecnico di Torino
//
// Modified work:
// Copyright (C) 2017  
// Vinh Dang (vqd8a@virginia.edu)
// University of Virginia
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public License
// as published by the Free Software Foundation; either version 2
// of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.

#ifdef DEVICE_EMU_DBG
#include <assert.h>
#include <stdio.h>
#endif

#include "common.h"
#include "gpu_functions.h"

#define myId		    threadIdx.x
#define thread_count    blockDim.x
#define nstreams        gridDim.x

extern __shared__ ST_BLOCK shared_base[];
__global__ void nfa_kernel(	st_t *nfa_tables,
							st_t *src_tables,
							unsigned int *input_transition_tables,
							symbol_fetch *input,
							unsigned long *cur_size_vec,
							ST_BLOCK *_svs,// ST_BLOCK is just a fancy name for a chunk of 32-bit unsigned data
							unsigned int *st_vec_lengths,
							ST_BLOCK *persistents,
							unsigned int *match_count, match_type *match_array, unsigned int match_vec_size,
							unsigned int *accum_nfa_table_lengths, unsigned int *accum_offset_table_lengths, unsigned int *accum_state_vector_lengths){
	
	__shared__ unsigned int shr_match_count;//Note: initializing is not allowed for shared variable
	shr_match_count = 0;
	
	unsigned int tmp_match_count;
	
	match_type tmp_match;
	
	// cur_size is the input string length
	size_t cur_size = (blockIdx.x == 0 ? cur_size_vec[blockIdx.x] : cur_size_vec[blockIdx.x] - cur_size_vec[blockIdx.x - 1]);

	// skip to the right input string
	if(blockIdx.x > 0)
		input += (cur_size_vec[blockIdx.x - 1]/fetch_bytes); 

	// get the right final_vector for reading the initial state and
	// storing the final output
	unsigned int st_vec_length             = st_vec_lengths            [blockIdx.y];
	unsigned int accum_state_vector_length = accum_state_vector_lengths[blockIdx.y]; 
	unsigned int accum_nfa_table_length    = accum_nfa_table_lengths   [blockIdx.y];
	unsigned int accum_offset_table_length = accum_offset_table_lengths[blockIdx.y];
	
	ST_BLOCK *final_vector = _svs + st_vec_length*blockIdx.x + accum_state_vector_length*nstreams;

	// shared_base points to the beginning of shared memory
	ST_BLOCK *status_vector = &shared_base[0];	
	ST_BLOCK *future_status_vector = shared_base + st_vec_length;

	//Copy the initial status vector from global to shared to set the input state
	for(unsigned int j = myId; j < st_vec_length; j += thread_count)
		//final_vector here is a misnomer as it is set with the initial state bit enabled
		status_vector[j] = final_vector[j];
	__syncthreads();

	unsigned int limit = cur_size; //printf("cur_size %d\n",cur_size);

	//Payload loop
	for(unsigned int p=0; p<limit; p+=fetch_bytes, input++){	
		symbol_fetch Input_ = *input;//fetch 4 bytes from the input string
		for (unsigned int byt = 0; byt < fetch_bytes; byt++) {
			unsigned int Input = Input_ & 0xFF;//extract 1 byte
			Input_  = Input_ >> 8;//Input_ right-shifted by 8 bits
			
			// input_transition_table contains the cumulative number of transitions for each input symbol
			unsigned int tr_base   = input_transition_tables[Input   + accum_offset_table_length];
			unsigned int tr_number = input_transition_tables[Input+1 + accum_offset_table_length] - tr_base;
			
			// Reset the future status vector
			// Persistent (self-loop'd) states are never reset once reached.
#pragma unroll 2
			for(unsigned w = myId; w < st_vec_length; w += thread_count)
				//future_status_vector[w] = persistents[w + accum_state_vector_length] & status_vector[w];
				future_status_vector[w] = 0;//might work too, since persistents vector is not used
			__syncthreads();

			for(unsigned int i=myId; i<tr_number; i+=thread_count) {
				// Each thread reads 1 transition at each step.
				st_t dst_state = nfa_tables[i + tr_base + accum_nfa_table_length];
				st_t src_state = src_tables[i + tr_base + accum_nfa_table_length];  
		
// These macros are there to extract the relevant fields.
// Bits and chunks are there to select the right bit in the state vectors.
#define src_bit  (1 << (src_state % bit_sizeof(ST_BLOCK)))
#define dst_bit  (1 << (dst_state % bit_sizeof(ST_BLOCK)))
#define src_chunk (src_state / bit_sizeof(ST_BLOCK))
#define dst_chunk (dst_state / bit_sizeof(ST_BLOCK))

				ST_BLOCK lo_block = src_bit & status_vector[src_chunk];
				if(lo_block) {
					if (dst_state < 0) {//Added for matching operation: check if the dst state is an accepting state
						dst_state = -dst_state;
						tmp_match_count = atomicAdd(&shr_match_count, 1);//printf("Inside kernel-low, offset: %d, state: %d, count %d\n",p, dst_state, shr_match_count);
						//match_offset[match_vec_size*blockIdx.x + shr_match_count-1 + blockIdx.y*match_vec_size*nstreams] = p + byt;
						//match_states[match_vec_size*blockIdx.x + shr_match_count-1 + blockIdx.y*match_vec_size*nstreams] = dst_state;
						tmp_match.off = p + byt;
						tmp_match.stat= dst_state;
						match_array[tmp_match_count + match_vec_size*(blockIdx.x + blockIdx.y*nstreams)] = tmp_match;
					}
					atomicOr(&future_status_vector[dst_chunk], dst_bit);    //unsigned int atomicOr(unsigned int* address, unsigned int val);
				}
			}
			// Swap status_vector and future_status_vector
			if(status_vector == shared_base){
				status_vector = shared_base + st_vec_length;
				future_status_vector = shared_base;
			} else {
				status_vector = shared_base;
				future_status_vector = shared_base + st_vec_length;
			}
			__syncthreads();
		}
	}

	//Copy the result vector from shared to device memory
#pragma unroll
	for(unsigned int j = myId; j < st_vec_length; j += thread_count) {
		final_vector[j] = status_vector[j];
	}	
	match_count[blockIdx.x + blockIdx.y*gridDim.x] = shr_match_count;
	
	__syncthreads();
}
